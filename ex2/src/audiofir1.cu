#include "hip/hip_runtime.h"
#include "audiofir.hpp"

#include <cassert>

#include <hip/hip_runtime_api.h>

__global__
static void audiofir_kernel(float *yout, float *yin,
    float *coeff, int n, int len)
{
    assert(yout != nullptr);
    assert(yin != nullptr);
    assert(coeff != nullptr);
    assert(n < len);

    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < len)
    {
        return;
    }

    auto j = i;
    auto sum = 0.0f;
    for(auto k = 0; k <= n; ++k, --j)
    {
        if(j < 0)
        {
            // For non-existent samples, tract them as zero
            // So there is no need to make further sums
            break;
        }

        sum += (yin[j] * coeff[k]);
    }

    yout[i] = sum;
}

void audiofir(float *yout, float *yin,
    float *coeff, int n, int len)
{
    checkCudaErrors(hipSetDevice(0));

    float* d_yout;
    float* d_yin;
    float* d_coeff;

    checkCudaErrors(hipMalloc(&d_yout, sizeof(float) * 2 * len));
    checkCudaErrors(hipMalloc(&d_yin, sizeof(float) * 2 * len));
    checkCudaErrors(hipMalloc(&d_coeff, sizeof(float) * (n + 1)));

    assert(d_yout != nullptr);
    assert(d_yin != nullptr);
    assert(d_coeff != nullptr);

    checkCudaErrors(hipMemcpy(d_yin, yin, sizeof(float) * 2 * len, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_coeff, coeff, sizeof(float) * (n + 1), hipMemcpyHostToDevice));

    hipEvent_t start, stop; // pomiar czasu wykonania jądra
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));

    constexpr auto K = 512;
    audiofir_kernel<<<(len + K - 1) / K, K>>>(d_yout, d_yin, d_coeff, n, len);
    audiofir_kernel<<<(len + K - 1) / K, K>>>(d_yout+len, d_yin+len, d_coeff, n, len);
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));

    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(yout, d_yout, sizeof(float) * 2 * len, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_yin));
    checkCudaErrors(hipFree(d_yout));
    checkCudaErrors(hipFree(d_coeff));

    checkCudaErrors(hipDeviceReset());

    printf("GPU (kernel) time = %.3f ms (%6.3f GFLOP/s)\n",
            elapsedTime, 1e-6 * 2*((double)n+1) * 2*((double)len) / elapsedTime);
}

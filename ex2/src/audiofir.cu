#include "hip/hip_runtime.h"
#include "audiofir.hpp"

#include <cassert>

#include <hip/hip_runtime_api.h>

__global__
static void audiofir_kernel(float *yout, const float *yin,
    const float *coeff, int n)
{
    assert(yout != nullptr);
    assert(yin != nullptr);
    assert(coeff != nullptr);

    const int i = threadIdx.x + blockIdx.x * blockDim.x;
	yin += i;
	yout += i;

    float sum = 0.0f;
    for(int k = 0; k <= n; ++k)
    {
        const float yin_elem = *(yin--);
        const float coeff_elem = *(coeff++);
        sum += (yin_elem * coeff_elem);
    }

    *yout = sum;
}

void audiofir(float *yout, const float *yin,
    const float *coeff, int n, int len)
{
    checkCudaErrors(hipSetDevice(0));

    float* d_yout;
    float* d_yin;
    float* d_coeff;

    static constexpr int K = 512;
    const unsigned int len_1 = (K * ((len + K - 1) / K));
    printf("%d %d\n", len, len_1);

    checkCudaErrors(hipMalloc(&d_yout, sizeof(float) * 2 * len_1));
    checkCudaErrors(hipMalloc(&d_yin, sizeof(float) * 2 * (len_1 + n)));
    checkCudaErrors(hipMalloc(&d_coeff, sizeof(float) * (n + 1)));

    assert(d_yout != nullptr);
    assert(d_yin != nullptr);
    assert(d_coeff != nullptr);

	checkCudaErrors(hipMemset(d_yin, 0, sizeof(float) * 2 * (len_1 + n)));
    checkCudaErrors(hipMemcpy(d_yin + n, yin, sizeof(float) * len, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_yin + n + len_1 + n, yin + len, sizeof(float) * len, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_coeff, coeff, sizeof(float) * (n + 1), hipMemcpyHostToDevice));

    checkCudaErrors(hipDeviceSynchronize());

    hipEvent_t start, stop; // pomiar czasu wykonania jądra
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));

    audiofir_kernel<<<(len + K - 1) / K, K>>>(d_yout, d_yin + n, d_coeff, n);
    audiofir_kernel<<<(len + K - 1) / K, K>>>(d_yout + len_1, d_yin + n + len_1 + n, d_coeff, n);
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));

    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(yout, d_yout, sizeof(float) * len, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(yout+len, d_yout+len_1, sizeof(float) * len, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_yin));
    checkCudaErrors(hipFree(d_yout));
    checkCudaErrors(hipFree(d_coeff));

    checkCudaErrors(hipDeviceReset());

    printf("GPU (kernel) time = %.3f ms (%6.3f GFLOP/s)\n",
            elapsedTime, 1e-6 * 2*((double)n+1) * 2*((double)len) / elapsedTime);
}

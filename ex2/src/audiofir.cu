#include "hip/hip_runtime.h"
#include "audiofir.hpp"

#include <cassert>

#include <hip/hip_runtime_api.h>

static constexpr int N = 1024; // maksymalny rząd filtru FIR 
static constexpr int K = 512; // Ilość wątków w bloku

__constant__ static float fir_coeff[N + 1];

__global__
static void audiofir_kernel(float *yout, const float *yin, int n)
{
    assert(yout != nullptr);
    assert(yin != nullptr);
	assert(n <= N);
	assert(threadIdx.x < K);

	__shared__ float ytile[N + K];

    const int i = threadIdx.x + blockIdx.x * blockDim.x;
	for (int j = i; j >= (i - n); j -= K)
	{
		ytile[j] = yin[i];
	}



	yin += i;
	yout += i;

	float* coeff = fir_coeff;
    float sum = 0.0f;
    for(int k = 0; k <= n; ++k)
    {
        const float yin_elem = *(yin--);
        const float coeff_elem = *(coeff++);
        sum += (yin_elem * coeff_elem);
    }

    *yout = sum;
}

void audiofir(float *yout, const float *yin,
    const float *coeff, int n, int len)
{
	assert(n <= N);

    checkCudaErrors(hipSetDevice(0));

    float* d_yout;
    float* d_yin;

    const unsigned int len_1 = (K * ((len + K - 1) / K));

    checkCudaErrors(hipMalloc(&d_yout, sizeof(float) * 2 * len_1));
    checkCudaErrors(hipMalloc(&d_yin, sizeof(float) * 2 * (len_1 + n)));

    assert(d_yout != nullptr);
    assert(d_yin != nullptr);

	checkCudaErrors(hipMemset(d_yin, 0, sizeof(float) * 2 * (len_1 + n)));
    checkCudaErrors(hipMemcpy(d_yin + n, yin, sizeof(float) * len, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_yin + n + len_1 + n, yin + len, sizeof(float) * len, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(fir_coeff), coeff, sizeof(float) * (n + 1)));

    checkCudaErrors(hipDeviceSynchronize());

    hipEvent_t start, stop; // pomiar czasu wykonania jądra
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));

    audiofir_kernel<<<(len + K - 1) / K, K>>>(d_yout, d_yin + n, n);
    audiofir_kernel<<<(len + K - 1) / K, K>>>(d_yout + len_1, d_yin + n + len_1 + n, n);
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));

    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(yout, d_yout, sizeof(float) * len, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(yout+len, d_yout+len_1, sizeof(float) * len, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_yin));
    checkCudaErrors(hipFree(d_yout));

    checkCudaErrors(hipDeviceReset());

    printf("GPU (kernel) time = %.3f ms (%6.3f GFLOP/s)\n",
            elapsedTime, 1e-6 * 2*((double)n+1) * 2*((double)len) / elapsedTime);
}

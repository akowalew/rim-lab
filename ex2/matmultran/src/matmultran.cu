#include "hip/hip_runtime.h"
#include "matmultran.hpp"

#include <cassert>

#include <hip/hip_runtime_api.h>

#define K 16 // rozmiar "kafelka"

__global__
static void matmultran_kernel(float *C, float *A, int m, int n)
{
    int tx = threadIdx.x; // kolumna wątku w ramach "kafelka"
    int ty = threadIdx.y; // wiersz wątku w ramach "kafelka"
    int ix = blockIdx.x * K + tx; // kolumna wątku w sieci
    int iy = blockIdx.y * K + ty; // wiersz wątku w sieci
    int iAT = blockIdx.x * K * n; // początek "kafelka" w A
    int iA = blockIdx.y * K * n; // początek "kafelka" w AT
    float s = 0;

    __shared__ float As[K][K], ATs[K][K];
    for(int t = 0; t < n / K; t++, iA += K, iAT += K)
    {
        As [ty][tx] = A[iA + ty*n + tx];
        ATs[ty][tx] = A[iAT + tx*n + ty];

        __syncthreads();

        #pragma unroll
        for (int k = 0; k < K; k++)
        {
            s += As[ty][k] * ATs[k][tx];
        }

        __syncthreads();
    }

    C[iy*m + ix] = s;
}

void matmultran(float *C, float *A, int m, int n)
{
    checkCudaErrors(hipSetDevice(0));

    float *dev_A, *dev_C;
    checkCudaErrors(hipMalloc(&dev_A, m*n*sizeof(float)));
    checkCudaErrors(hipMalloc(&dev_C, m*m*sizeof(float)));
    checkCudaErrors(hipMemcpy(dev_A, A, m*n*sizeof(float), hipMemcpyHostToDevice));

    dim3 dimGrid(m/K, m/K), dimBlock(K, K);

    hipEvent_t start, stop; // pomiar czasu wykonania jądra
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipEventRecord(start, 0));
    matmultran_kernel<<<dimGrid, dimBlock>>>(dev_C, dev_A, m, n);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipEventRecord(stop, 0));

    checkCudaErrors(hipEventSynchronize(stop));

    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&elapsedTime,start, stop));

    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(C, dev_C, m*m*sizeof(float), hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(dev_C));
    checkCudaErrors(hipFree(dev_A));

    checkCudaErrors(hipDeviceReset()); // dla debuggera

    printf("GPU (kernel) time = %.3f ms (%6.3f GFLOP/s)\n", elapsedTime, 2e-6 * m * m * n / elapsedTime);
}

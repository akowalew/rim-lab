#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h> //-I$(NVCUDASAMPLES_ROOT)/common/inc
#include <hipblas.h>   // wymaga konsolidacji z cublas.lib

#include <stdio.h>

#include "raindemo.hpp"

#define TRY(code) checkCudaErrors((hipError_t) (code))

void csmp(float *xs, float *y, float *A, int N, int M, ...)
{
	float *dev_A, *dev_r, *dev_sp;
	hipblasHandle_t h;
	float one = 1.0f, zero = 0.0f, nrm2y, nrm2a, nrm2r, s;
	int i, t;

	TRY(hipSetDevice(0));

	TRY(hipMalloc(&dev_A, M*N * sizeof(float)));
	TRY(hipMalloc(&dev_r, M * sizeof(float)));
	TRY(hipMalloc(&dev_sp, N * sizeof(float)));

	TRY(hipblasCreate(&h));

	// TODO: prolog algorytmu

	for (t = 1, nrm2r = nrm2y; t <= 50 && nrm2r > 0.05*nrm2y; t++)
	{
		// TODO: iteracja algorytmu

		printf("iter.%3d: x(%3d) <- %4.2f, nrm2res=%4.2f\n",
			t, i, s, nrm2r);
	}

	TRY(hipblasDestroy(h));

	TRY(hipFree(dev_sp));
	TRY(hipFree(dev_r));
	TRY(hipFree(dev_A));

	TRY(hipDeviceReset()); // dla debuggera i profilera
}
